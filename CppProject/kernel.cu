
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
using namespace std;

extern "C" 
{
	__global__ void addKernel(int* a, int* b, int* c)
	{
		int i = blockDim.x * blockIdx.x + threadIdx.x;
		c[i] = a[i] + b[i];
	}
}