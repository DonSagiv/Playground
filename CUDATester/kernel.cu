
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>
using namespace std;

__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{

}
